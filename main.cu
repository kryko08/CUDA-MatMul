#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"


#define CUDA_CHECK_RETURN( value ) {                            \
    hipError_t err = value;                                    \
    if( err != hipSuccess ) {                                  \
        fprintf( stderr, "Error %s at line %d in file %s\n",    \
                hipGetErrorString(err), __LINE__, __FILE__ );  \
        exit( 1 );                                              \
    } }

#define BLOCK_SIZE 16 // 16 * 16 = 256 threads in single block 

__global__ void matMul(int* Avalues, int* Bvalues, int* Out, int Awidth, int Bwidth){
    int value = 0;
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    for (int i=0; i < Awidth; ++i){
        value += Avalues[row * Awidth + i] * Bvalues[i * Bwidth + col];
    }
    Out[row * Bwidth + col] = value;
}

int main(){
    int width_a;
    int height_a;
    printf("Input matrix A width. Must be multiple of 16 (Block size).\n");
    scanf("%d", &width_a);
    printf("Input matrix  A height. Must be multiple of 16.\n");
    scanf("%d", &height_a);
    
    // Suppose that FirstMatrix width = SecondMatrix height and FirstMatrix height = SecondMatrix width
    int A_height = height_a;
    int A_width = width_a;
    int *A_values = (int *)malloc(sizeof(int) * A_width * A_height);

    int B_height = height_a;
    int B_width = width_a;
    int *B_values = (int *)malloc(sizeof(int) * B_width * B_height);

    int C_height = A_height;
    int C_width = B_width;
    int *C_values = (int *)malloc(sizeof(int) * C_width * C_height);

    int *d_A_values;
    int *d_B_values;
    int *d_C_values;

    // initialize matrices values 
    for (int i = 0; i < A_width * A_width; ++i){
        A_values[i] = i;
    }

    for (int i = 0; i < B_width * B_height; ++i){
        B_values[i] = i;
    }

    // Allocate on CUDA
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_A_values, sizeof(int) * A_height * A_width));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_B_values, sizeof(int) * B_height * B_width));
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_C_values, sizeof(int) * C_height * C_width));

    // Copy values to CUDA 
    hipMemcpy(d_A_values, A_values, sizeof(int) * A_height * A_width, hipMemcpyHostToDevice);
    hipMemcpy(d_B_values, B_values, sizeof(int) * B_height * B_width, hipMemcpyHostToDevice);

    
    // Grid and Block dimension setup
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(C_width / dimBlock.x, C_height / dimBlock.y);
    matMul<<<dimGrid, dimBlock>>>(d_A_values, d_B_values, d_C_values, A_width, B_width);  // Run Kernel

    // Copy values from device to host
    hipMemcpy(C_values, d_C_values, sizeof(int) * C_width * C_height, hipMemcpyDeviceToHost);

    // print values 
    for (int i = 0; i < C_width * C_height; ++i){
        printf("%d\n", C_values[i]);
    }
    return 0;
}

